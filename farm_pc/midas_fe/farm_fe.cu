#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <unistd.h>

#include "midas.h"
#include "odbxx.h"
#include "msystem.h"
#include "mcstd.h"
#include "experim.h"
#include "switching_constants.h"
#include "link_constants.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <sstream>
#include <fstream>

#include "mudaq_device.h"
#include "mfe.h"

using namespace std;
using midas::odb;

/*-- Globals -------------------------------------------------------*/

/* The frontend name (client name) as seen by other MIDAS clients   */
const char *frontend_name = "Stream Frontend";

/* The frontend file name, don't change it */
const char *frontend_file_name = __FILE__;

/* frontend_loop is called periodically if this variable is TRUE    */
BOOL frontend_call_loop = FALSE;

/* a frontend status page is displayed with this frequency in ms */
INT display_period = 0;

/* DMA Buffer and related */
volatile uint32_t *dma_buf;
size_t dma_buf_size = MUDAQ_DMABUF_DATA_LEN;
uint32_t dma_buf_nwords = dma_buf_size/sizeof(uint32_t);
uint32_t laddr;
uint32_t newdata;
uint32_t readindex;
uint32_t wlen;
uint32_t lastreadindex;
uint32_t lastlastWritten;
uint32_t lastRunWritten;
bool moreevents;
bool firstevent;

/* maximum event size produced by this frontend */
INT max_event_size = dma_buf_size;

/* maximum event size for fragmented events (EQ_FRAGMENTED) */
INT max_event_size_frag = 5 * 1024 * 1024;

/* buffer size to hold events */
INT event_buffer_size = 32 * max_event_size;

mudaq::DmaMudaqDevice * mup;
mudaq::DmaMudaqDevice::DataBlock block;

/*-- Function declarations -----------------------------------------*/

INT frontend_init();
INT frontend_exit();
INT begin_of_run(INT run_number, char *error);
INT end_of_run(INT run_number, char *error);
INT pause_run(INT run_number, char *error);
INT resume_run(INT run_number, char *error);
INT frontend_loop();

INT read_stream_event(char *pevent, INT off);
INT read_stream_thread(void *param);

INT poll_event(INT source, INT count, BOOL test);
INT interrupt_configure(INT cmd, INT source, POINTER_T adr);


void setup_odb();
void setup_watches();

INT init_mudaq();

void link_active_settings_changed(odb);
void datagen_settings_changed(odb);
/*-- Equipment list ------------------------------------------------*/

EQUIPMENT equipment[] = {

   {"Stream",                /* equipment name */
    {1, 0,                   /* event ID, trigger mask */
     "SYSTEM",               /* event buffer */
     EQ_USER,                /* equipment type */
     0,                      /* event source crate 0, all stations */
     "MIDAS",                /* format */
     TRUE,                   /* enabled */
     RO_RUNNING  | RO_STOPPED | RO_ODB,             /* read only when running */
     100,                    /* poll for 100ms */
     0,                      /* stop run after this event limit */
     0,                      /* number of sub events */
     0,                      /* don't log history */
     "", "", "",},
     NULL,                    /* readout routine */
    },

   {""}
};

/*-- Frontend Init -------------------------------------------------*/

INT frontend_init()
{
    // TODO: for debuging
    odb::set_debug(true);

    set_equipment_status(equipment[0].name, "Initializing...", "var(--myellow)");

    // setup odb and watches
    setup_odb();
    setup_watches();

    // init dma and mudaq device
    INT status = init_mudaq();
    if (status != SUCCESS) return FE_ERR_DRIVER;

    usleep(5000);

    // create ring buffer for readout thread
    create_event_rb(0);

    // create readout thread
    ss_thread_create(read_stream_thread, NULL);

    set_equipment_status(equipment[0].name, "Ready for running", "var(--mgreen)");

    //Set our transition sequence. The default is 500.
    cm_set_transition_sequence(TR_START,300);

    //Set our transition sequence. The default is 500. Setting it
    // to 700 means we are called AFTER most other clients.
    cm_set_transition_sequence(TR_STOP,700);

    return SUCCESS;
}

// ODB Setup //////////////////////////////
void setup_odb(){

    // Map /equipment/Stream/Settings
    odb datagen_settings = {
        {"Divider", 1000},     // int
        {"Enable", false},     // bool
    };

    datagen_settings.connect("/Equipment/Stream/Settings/Datagenerator", true);

    odb dma_settings = {
            {"dma_buf_nwords", int(dma_buf_nwords)},
            {"dma_buf_size", int(dma_buf_size)},
    };

    dma_settings.connect("/Equipment/Stream/Settings/DMA_Settings", true);

    // add custom page to ODB
    odb custom("/Custom");
    custom["Farm&"] = "farm.html";

}

void setup_watches(){

    // datagenerator changed settings
    odb datagen("/Equipment/Stream/Settings/Datagenerator");
    datagen.watch(datagen_settings_changed);

    // link mask changed settings
    odb links("/Equipment/Links/Settings/LinkMask");
    links.watch(link_active_settings_changed);

}

void datagen_settings_changed(odb o)
{
    std::string name = o.get_name();

    std::cout << name << std::endl;

    if (name == "Divider") {
        bool value = o;
        cm_msg(MINFO, "datagen_settings_changed", "Set Divider to %d", value);
        mup->write_register_wait(DATAGENERATOR_DIVIDER_REGISTER_W, o, 100);
        // TODO: test me
    }

    if (name == "Enable") {
        bool value = o;
        cm_msg(MINFO, "datagen_settings_changed", "Set Disable to %d", value);
        //this is set once we start the run
    }

}

void link_active_settings_changed(odb o){

    /* get link active from odb */
    uint64_t link_active_from_odb = 0;
    //printf("Data link active: 0x");
    int idx=0;
    for(int link : o) {
        int offset = 0;//MAX_LINKS_PER_SWITCHINGBOARD* switch_id;
        if(link & FEBLINKMASK::DataOn)
            //a standard FEB link (SC and data) is considered enabled if RX and TX are.
            //a secondary FEB link (only data) is enabled if RX is.
            //Here we are concerned only with run transitions and slow control, the farm frontend may define this differently.
            link_active_from_odb += (1 << idx);
        //printf("%u",(frontend_board_active_odb[offset + link] & FEBLINKMASK::DataOn?1:0));
        idx ++;
    }
    //printf("\n");
    //mup->write_register(DATA_LINK_MASK_REGISTER_HIGH_W, enablebits >> 32); TODO make 64 bits
    mup->write_register(DATA_LINK_MASK_REGISTER_W, link_active_from_odb & 0xFFFFFFFF);

}

// INIT MUDAQ //////////////////////////////
INT init_mudaq(){

    // Allocate memory for the DMA buffer - this can fail!
    if(hipHostMalloc( (void**)&dma_buf, dma_buf_size ) != hipSuccess){
        cout << "Allocation failed, aborting!" << endl;
        cm_msg(MERROR, "frontend_init" , "Allocation failed, aborting!");
        return FE_ERR_DRIVER;
    }

    // initialize to zero
    for (int i = 0; i < dma_buf_nwords ; i++) {
        (dma_buf)[i] = 0;
    }

    // open mudaq
    mup = new mudaq::DmaMudaqDevice("/dev/mudaq0");
    if ( !mup->open() ) {
        cout << "Could not open device " << endl;
        cm_msg(MERROR, "frontend_init" , "Could not open device");
        return FE_ERR_DRIVER;
    }

    // check mudaq
    if ( !mup->is_ok() )
        return FE_ERR_DRIVER;
    else {
        cm_msg(MINFO, "frontend_init" , "Mudaq device is ok");
    }

    // set fpga write pointers
    lastlastWritten = 0;
    lastRunWritten = mup->last_written_addr();

    // map memory to bus addresses for FPGA
    struct mesg user_message;
    user_message.address = dma_buf;
    user_message.size = dma_buf_size;

    int ret_val = mup->map_pinned_dma_mem( user_message );
    
    if (ret_val < 0) {
        cout << "Mapping failed " << endl;
        cm_msg(MERROR, "frontend_init" , "Mapping failed");
        mup->disable();
        mup->close();
        free( (void *)dma_buf );
        delete mup;
        return FE_ERR_DRIVER;
    }

    // switch off and reset DMA for now
    mup->disable();
    usleep(2000);

    // switch off the data generator (just in case ..)
    mup->write_register(DATAGENERATOR_REGISTER_W, 0x0);
    usleep(2000);

    // DMA_CONTROL_W
    mup->write_register(0x5,0x0);

    //set data link enable
    odb link;
    link.connect("/Equipment/Links/Settings/LinkMask");
    link_active_settings_changed(link);

    return SUCCESS;
}


INT db_watch_datagen_thread(void *param){

    midas::odb datagen("/Equipment/Stream/Settings/Datagenerator");
    datagen.watch([](midas::odb &o) {
        std::cout << "Value of key \"" + o.get_full_path() + "\" changed to " << o << std::endl;

        if (o.get_full_path() == "Enable") {
            //this is set once we start the run
        }

        if (o.get_full_path() == "Divider") {
            mup->write_register_wait(DATAGENERATOR_DIVIDER_REGISTER_W, o, 100);
        }
    });

    do {
        int status = cm_yield(100);
        if (status == SS_ABORT || status == RPC_SHUTDOWN)
            break;
    } while (!ss_kbhit());

    return 1;
}

/*-- Frontend Exit -------------------------------------------------*/

INT frontend_exit()
{
   if (mup) {
      mup->disable();
      mup->close();
      delete mup;
   }

   // following code crashes the frontend, please fix!
   // free( (void *)dma_buf );
   hipHostFree((void *)dma_buf);
   
   return SUCCESS;
}


/*-- Begin of Run --------------------------------------------------*/

INT begin_of_run(INT run_number, char *error)
{ 
   set_equipment_status(equipment[0].name, "Starting run", "var(--myellow)");
   
   mudaq::DmaMudaqDevice & mu = *mup;
   
   // Reset last written address used for polling
   laddr = mu.last_written_addr();
   newdata = 0;
   readindex = 0;
   moreevents = false;
   firstevent = true;

   // reset all
   uint32_t reset_reg = 0;
   reset_reg |= 1<<RESET_BIT_EVENT_COUNTER;
   reset_reg |= 1<<RESET_BIT_DATAGEN;
   mu.write_register_wait(RESET_REGISTER_W, reset_reg, 100);

   // empty dma buffer
   for (int i = 0; i < dma_buf_nwords ; i++) {
      (dma_buf)[i] = 0;
   }

   // Enable register on FPGA for continous readout and enable dma
   mu.enable_continous_readout(0);
   usleep(10);
   mu.write_register_wait(RESET_REGISTER_W, 0x0, 100);

   // Set up data generator: enable only if set in ODB
   uint32_t reg=mu.read_register_rw(DATAGENERATOR_REGISTER_W);
   odb datagen_settings;
   datagen_settings.connect("/Equipment/Stream/Settings/Datagenerator");
   if(datagen_settings["Enable"]) {
       // TODO: test me
       mu.write_register_wait(DMA_SLOW_DOWN_REGISTER_W, datagen_settings["Divider"], 100);
       reg = SET_DATAGENERATOR_BIT_ENABLE(reg);
   }
   mu.write_register(DATAGENERATOR_REGISTER_W,reg);

   // reset lastlastwritten
   lastlastWritten = 0;
   lastRunWritten = mu.last_written_addr();//lastWritten;

   // Note: link masks are already set during fe_init and via ODB callback

   set_equipment_status(equipment[0].name, "Running", "var(--mgreen)");
   
   return SUCCESS;
}

/*-- End of Run ----------------------------------------------------*/

INT end_of_run(INT run_number, char *error)
{

   mudaq::DmaMudaqDevice & mu = *mup;
   printf("farm_fe: Waiting for buffers to empty\n");
   uint16_t timeout_cnt = 0;
   while(! mu.read_register_ro(BUFFER_STATUS_REGISTER_R) & 1<<0/* TODO right bit */ &&
         timeout_cnt++ < 50) {
      printf("Waiting for buffers to empty %d/50\n", timeout_cnt);
      timeout_cnt++;
      usleep(1000);
   };

   if(timeout_cnt>=50) {
      cm_msg(MERROR,"farm_fe","Buffers on Switching Board not empty at end of run");
      set_equipment_status(equipment[0].name, "Not OK", "var(--mred)");
      //return CM_TRANSITION_CANCELED;
   }else{
      printf("Buffers all empty\n");
   }

   // TODO: Find a better way to see when DMA is finished.

   printf("Waiting for DMA to finish\n");
   usleep(1000); // Wait for DMA to finish
   timeout_cnt = 0;
   while(mu.last_written_addr() != lastlastWritten && //(readindex % dma_buf_nwords) &&
         timeout_cnt++ < 50) {
      printf("Waiting for DMA to finish %d/50\n", timeout_cnt);
      timeout_cnt++;
      usleep(1000);
   };

   if(timeout_cnt>=50) {
      cm_msg(MERROR,"farm_fe","DMA did not finish");
      set_equipment_status(equipment[0].name, "Not OK", "var(--mred)");
//      return CM_TRANSITION_CANCELED;
   }else{
      printf("DMA is finished\n");
   }

    // stop generator
   uint32_t datagen_setup = 0;
   datagen_setup = UNSET_DATAGENERATOR_BIT_ENABLE(datagen_setup);
   mu.write_register_wait(DATAGENERATOR_REGISTER_W, datagen_setup, 100);
   mu.write_register_wait(DMA_SLOW_DOWN_REGISTER_W, 0x0, 100);

   // disable DMA
   mu.disable();

   set_equipment_status(equipment[0].name, "Ready for running", "var(--mgreen)");
   
   return SUCCESS;
}

/*-- Pause Run -----------------------------------------------------*/

INT pause_run(INT run_number, char *error)
{
   mudaq::DmaMudaqDevice & mu = *mup;
   
//   uint32_t datagen_setup = mu.read_register_rw(DATAGENERATOR_REGISTER_W);
//   datagen_setup = UNSET_DATAGENERATOR_BIT_ENABLE(datagen_setup);
//   mu.write_register_wait(DATAGENERATOR_REGISTER_W, datagen_setup,1000);

   // disable DMA
   mu.disable(); // Marius Koeppel: not sure if this works
   
   set_equipment_status(equipment[0].name, "Paused", "var(--myellow)");
   
   return SUCCESS;
}

/*-- Resume Run ----------------------------------------------------*/

INT resume_run(INT run_number, char *error)
{
   mudaq::DmaMudaqDevice & mu = *mup;
   
//   uint32_t datagen_setup = mu.read_register_rw(DATAGENERATOR_REGISTER_W);
//   datagen_setup = SET_DATAGENERATOR_BIT_ENABLE(datagen_setup);
//   mu.write_register_wait(DATAGENERATOR_REGISTER_W, datagen_setup,1000);

   // enable DMA
   mu.enable_continous_readout(0); // Marius Koeppel: not sure if this works
   
   set_equipment_status(equipment[0].name, "Running", "var(--mgreen)");
   
   return SUCCESS;
}

/*-- Frontend Loop -------------------------------------------------*/

INT frontend_loop()
{
   /* if frontend_call_loop is true, this routine gets called when
      the frontend is idle or once between every event */
   return SUCCESS;
}

/*-- Trigger event routines ----------------------------------------*/

INT poll_event(INT source, INT count, BOOL test)
/* Polling routine for events. Returns TRUE if event
 is available. If test equals TRUE, don't return. The test
 flag is used to time the polling */
{
   /*
   if(moreevents && !test)
      return 1;
   
   mudaq::DmaMudaqDevice & mu = *mup;
   
   for (int i = 0; i < count; i++) {
      uint32_t addr = mu.last_written_addr();
      if ((addr != laddr) && !test) {
         if (firstevent) {
            newdata = addr;
            firstevent = false;
         } else {
            if(addr > laddr)
               newdata = addr - laddr;
            else
               newdata = 0x10000 - laddr + addr;
         }
         if (newdata > 0x10000) {
            return 0;
         }
         laddr = addr;
         return 1;
      }
   }
   */
   return 0;
}

/*-- Interrupt configuration ---------------------------------------*/

INT interrupt_configure(INT cmd, INT source, POINTER_T adr)
{
   return SUCCESS;
}

/*-- Event readout -------------------------------------------------*/

INT read_stream_event(char *pevent, INT off)
{
   /*
   bk_init(pevent);
   
   DWORD *pdata;
   uint32_t read = 0;
   bk_create(pevent, "HEAD", TID_DWORD, (void **)&pdata);
   
   for (int i =0; i < 8; i ++) {
      *pdata++ = dma_buf[(++readindex)%dma_buf_nwords];
      read++;
   }
   
   bk_close(pevent, pdata);
   newdata -= read;
   
   if (read < newdata && newdata < 0x10000)
      moreevents = true;
   else
      moreevents = false;
   
   return bk_size(pevent);
   */
   return 0;
}

// check if the event is good
template < typename T >
INT check_event(T* buffer, uint32_t idx) {
    EVENT_HEADER* eh = (EVENT_HEADER*)(buffer + idx);
    BANK_HEADER* bh = (BANK_HEADER*)(eh + 1);

    if ( eh->event_id != 0x1 ) {
        printf("Error: Wrong event id 0x%08X\n", eh->event_id);
        return -1;
    }
    if ( eh->trigger_mask != 0x0 ) {
        printf("Error: Wrong trigger_mask 0x%08X\n", eh->trigger_mask);
        return -1;
    }
    if ( bh->flags != 0x11 ) {
        printf("Error: Wrong flags 0x%08X\n", bh->flags);
        return -1;
    }
    
    uint32_t eventDataSize = eh->data_size; // bytes
    //printf("eventDataSize = %u bytes\n", eventDataSize);

    // offset bank relative to event data
    uint32_t bankOffset = 8; // bytes
    // iterate through banks
    while(true) {
        BANK32* b = (BANK32*)(&buffer[idx + 4 + bankOffset / 4]);
	//printf("bank: name = %4.4s, data_size = %u bytes, offset = %u bytes\n", b->name, b->data_size, bankOffset);
        bankOffset += sizeof(BANK32) + b->data_size; // bytes
        if(bankOffset > eventDataSize) { sleep(10); return -1; }
        if(bankOffset == eventDataSize) break;
        // TODO: uncomment for new bank format from firmware
//        bankOffset += b->data_size % 8;
    }

    return 0;
}

int copy_event(uint32_t* dst, volatile uint32_t* src) {
    // copy event header and global bank header to destination
    std::copy_n(src, sizeof(EVENT_HEADER) / 4 + sizeof(BANK_HEADER) / 4, dst);
    // get header for future asjustment
    EVENT_HEADER* eh = (EVENT_HEADER*)(dst);
    BANK_HEADER* bh = (BANK_HEADER*)(eh + 1);

    // start from first bank
    int src_i = 6, dst_i = 6;

    while(true) {
        // get bank
        BANK32* bank = (BANK32*)(src + src_i);
        // copy bank to dst
        std::copy_n((uint32_t*)bank, sizeof(BANK32) / 4 + bank->data_size / 4, dst + dst_i);
        // go to next bank
        src_i += sizeof(BANK32) / 4 + bank->data_size / 4;
        // TODO: uncomment for new bank format from firmware
//        src_i += b->data_size % 8;
        // insert empty word if needed in dst
        dst_i += sizeof(BANK32) / 4 + bank->data_size / 4;
        if(src_i >= sizeof(EVENT_HEADER) / 4 + eh->data_size / 4) break;
        // at this point we expect next bank
        if(bank->data_size % 8) {
            // insert padding word
            dst[dst_i] = 0xFFFFFFFF;
            dst_i += 1;
        }
    }

    // update data_size's
    bh->data_size = dst_i * 4 - sizeof(EVENT_HEADER) - sizeof(BANK_HEADER);
    eh->data_size = dst_i * 4 - sizeof(EVENT_HEADER);

    return dst_i;
}

INT update_equipment_status(int status, int cur_status, EQUIPMENT *eq)
{
    
    if ( status != DB_SUCCESS ) {
        set_equipment_status(eq[0].name, "Buffer ERROR", "var(--myellow)");
        return -1;
    }

    if ( cur_status != DB_SUCCESS ) {
        set_equipment_status(eq[0].name, "Running", "var(--mgreen)");
    }

    return DB_SUCCESS;

}

/*-- Event readout -------------------------------------------------*/

INT read_stream_thread(void *param) {
    // get mudaq
    mudaq::DmaMudaqDevice & mu = *mup;

    int cur_status = -1;

    // tell framework that we are alive
    signal_readout_thread_active(0, TRUE);

    // obtain ring buffer for inter-thread data exchange
    int rbh = get_event_rbh(0);

    uint32_t max_requested_words = dma_buf_nwords/2;
    // request to read dma_buffer_size/2 (count in blocks of 256 bits)
    mu.write_register_wait(0xC, max_requested_words / (256/32), 100);

    while (is_readout_thread_enabled()) {
        // don't readout events if we are not running
        if (run_state != STATE_RUNNING) {
            set_equipment_status(equipment[0].name, "Not running", "var(--myellow)");
            //ss_sleep(100);
            //TODO: signalling from main thread?
            continue;
        }

        set_equipment_status(equipment[0].name, "Running", "var(--mgreen)");

        // start dma
        mu.enable_continous_readout(0);

        // wait for requested data
        while ( (mu.read_register_ro(0x1C) & 1) == 0 ) {}

        // disable dma
        mu.disable();
        // and get lastWritten
        lastlastWritten = 0;
        uint32_t lastWritten = mu.last_written_addr();
//        printf("lastWritten = 0x%08X\n", lastWritten);

        // print dma_buf content
//        for ( int i = lastWritten - 0x100; i < lastWritten + 0x100; i++) {
//            if(i % 8 == 0) printf("[0x%08X]", i);
//            printf("  %08X", dma_buf[i]);
//            if(i % 8 == 7) printf("\n");
//        } printf("\n");

        // walk events to find end of last event
        if(lastWritten < lastlastWritten) lastWritten += dma_buf_nwords;
        uint32_t offset = lastlastWritten;
        while(true) {
            // check enough space for header
            if(offset + 4 > lastWritten) break;
//            printf("event: offset = 0x%08X, event_id = 0x%08X, data_size = 0x%08X\n", offset, dma_buf[offset % dma_buf_nwords], dma_buf[(offset + 3) % dma_buf_nwords]);
            uint32_t eventLength = 16 + dma_buf[(offset + 3) % dma_buf_nwords];
            if(eventLength > max_requested_words * 4) {
                printf("ERROR: (eventLength = 0x%08X) > max_event_size\n", eventLength);
                break;
            }
            // check enough space for data
            if(offset + eventLength / 4 > lastWritten) break;
	    if(check_event(dma_buf, offset) < 0) {
                printf("ERROR: bad event\n");
                break;
            }
            offset += eventLength / 4;
        }
//        printf("lastlastWritten = 0x%08X, offset = 0x%08X, lastWritten = 0x%08X\n", lastlastWritten, offset, lastWritten);
        if(offset > dma_buf_nwords) offset -= dma_buf_nwords;
        lastWritten = offset;

        // get midas buffer
        uint32_t* pdata = nullptr;
        int rb_status = rb_get_wp(rbh, (void**)&pdata, 0);
        if ( rb_status != DB_SUCCESS ) {
            printf("ERROR: rb_get_wp -> rb_status != DB_SUCCESS\n");
            lastlastWritten = lastWritten;
            continue;
        }

        // number of words written to midas buffer
        uint32_t wlen = 0;

        // copy midas buffer and adjust bank data_size to multiple of 8 bytes
        for(int src_i = 0, dst_i = 0; src_i < lastWritten;) {
            int nwords = copy_event(pdata + dst_i, dma_buf + src_i);
            src_i += 4 + dma_buf[src_i + 3] / 4;
            dst_i += nwords;
            wlen = dst_i;
        }
        lastlastWritten = lastWritten;

        // copy data to midas and increment wp of the midas buffer
        if(lastWritten < lastlastWritten) {
            // partial copy when wrap around
            copy_n(&dma_buf[lastlastWritten], dma_buf_nwords - lastlastWritten, pdata);
            wlen += dma_buf_nwords - lastlastWritten;
            lastlastWritten = 0;
        }
        if(lastWritten != lastlastWritten) {
            // complete copy
            copy_n(&dma_buf[lastlastWritten], lastWritten - lastlastWritten, pdata + wlen);
            wlen += lastWritten - lastlastWritten;
            lastlastWritten = lastWritten;
        }

        // update midas buffer
        rb_status = rb_increment_wp(rbh, wlen * 4); // in byte length
        if ( rb_status != DB_SUCCESS ) {
            printf("ERROR: rb_increment_wp -> rb_status != DB_SUCCESS\n");
        }

        cur_status = update_equipment_status(rb_status, cur_status, equipment);
    }

    // tell framework that we finished
    signal_readout_thread_active(0, FALSE);

    return 0;
}
