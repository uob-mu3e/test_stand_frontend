#include "hip/hip_runtime.h"
/**
 * open a mudaq device and measure DMA speed
 * use data generator from counter with 250 MHz clock
 *
 * @author      Dorothea vom Bruch <vombruch@physi.uni-heidelberg.de>
 *              adapted from code by Fabian Foerster and Qinhua Huang
 * @date        2015-01-22
 */

#include <iostream>
#include <unistd.h>
#include <chrono>
#include <stdio.h>
#include <sstream>
#include <limits>
#include <fstream>
#include <sys/mman.h>
#include <chrono>
#include <stdlib.h>
#include <cassert>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <fcntl.h>

#include "mudaq_device.h"

using namespace std;

void print_usage(){
    cout << "Usage: " << endl;
    cout << "       dmatest <use_data_gen> <stop_dma> <use_loop>" << endl;
}

int main(int argc, char *argv[])
{

    if(argc < 4){
        print_usage();
        return -1;
    }

    if(atoi(argv[2]) == 1) {
        /* Open mudaq device */
        mudaq::DmaMudaqDevice mu("/dev/mudaq0");
        if ( !mu.open() ) {
            cout << "Could not open device " << endl;
            return -1;
        }

        mu.disable();
        uint32_t datagen_setup = 0;
        datagen_setup = UNSET_DATAGENERATOR_BIT_ENABLE(datagen_setup);
        mu.write_register_wait(DATAGENERATOR_REGISTER_W, datagen_setup, 100);
        mu.write_register_wait(DATAGENERATOR_DIVIDER_REGISTER_W, 0x0, 100);
        mu.write_register_wait(DATA_LINK_MASK_REGISTER_2_W, 0x0, 100);
        mu.write_register_wait(DATA_LINK_MASK_REGISTER_W, 0x0, 100);
        mu.write_register_wait(SWB_READOUT_STATE_REGISTER_W, 0x0, 100);
        mu.close();
        return 0;
    }

    ofstream myfile;
    myfile.open("memory_content.txt");
    if ( !myfile ) {
      cout << "Could not open file " << endl;
      return -1;
    }

    myfile << "idx" << "\t" << "data" << endl;

//    system("echo machmalkeins | sudo -S ../../../common/kerneldriver/compactify.sh");
//    usleep(1000000);
//    system("echo machmalkeins | sudo -S ../../../common/kerneldriver/compactify.sh");
//    usleep(1000000);

    size_t dma_buf_size = MUDAQ_DMABUF_DATA_LEN;
    volatile uint32_t *dma_buf;
    size_t size = MUDAQ_DMABUF_DATA_LEN;
    uint32_t dma_buf_nwords = dma_buf_size/sizeof(uint32_t);

    hipError_t cuda_error = hipHostMalloc( (void**)&dma_buf, size );
    if(cuda_error != hipSuccess){
        cout << "Error: " << hipGetErrorString(cuda_error) << endl;
        cout << "Allocation failed!" << endl;
        return -1;
    }

    int fd = open("/dev/mudaq0_dmabuf", O_RDWR);
    if(fd < 0) {
        printf("fd = %d\n", fd);
        return EXIT_FAILURE;
    }
    dma_buf = (uint32_t*)mmap(nullptr, MUDAQ_DMABUF_DATA_LEN, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
    if(dma_buf == MAP_FAILED) {
        printf("mmap failed: dmabuf = %x\n", dma_buf);
        return EXIT_FAILURE;
    }

    // initialize to zero
    for (int i = 0; i <  size/sizeof(uint32_t) ; i++) {
      (dma_buf)[ i ] = 0;
    }

    /* Open mudaq device */
    mudaq::DmaMudaqDevice mu("/dev/mudaq0");
    if ( !mu.open() ) {
        cout << "Could not open device " << endl;
        return -1;
    }

    if ( !mu.is_ok() ) return -1;
    cout << "MuDaq is ok" << endl;

    struct mesg user_message;
    user_message.address = dma_buf;
    user_message.size = size;

    /* map memory to bus addresses for FPGA */
    int ret_val = 0;//mu.map_pinned_dma_mem( user_message );

    if ( ret_val < 0 ) {
        cout << "Mapping failed " << endl;
        mu.disable();
        mu.close();
        free( (void *)dma_buf );
        return ret_val;
    }

    // request data to read dma_buffer_size/2 (count in blocks of 256 bits) 
    uint32_t max_requested_words = dma_buf_nwords/64;
    cout << "request " << max_requested_words << endl;
    mu.write_register_wait(0xC, max_requested_words / (256/32), 100);

    // Enable register on FPGA for continous readout and enable dma
    uint32_t lastlastWritten = mu.last_written_addr();
    mu.enable_continous_readout(0);

    // use on link to readout
    if (atoi(argv[1]) == 1) {
        uint32_t USE_GEN_LINK_setup = 0;
        uint32_t USE_LINK_setup = 0;

        //mu.write_register(SWB_READOUT_STATE_REGISTER_W, 0x0);
        
        // readout only link 0 without merging
        //USE_GEN_LINK_setup = SET_USE_BIT_GEN_LINK(USE_GEN_LINK_setup);
        //USE_LINK_setup = SET_USE_BIT_LINK(USE_LINK_setup);
        //mu.write_register(SWB_READOUT_LINK_REGISTER_W, 0x0);
        //mu.write_register(SWB_READOUT_STATE_REGISTER_W, USE_LINK_setup);
        
        // setup datagenerator

        mu.write_register(0x13, 0x9);
        mu.write_register(DATAGENERATOR_DIVIDER_REGISTER_W, 0x0);//3E8, 100);//3E8); // slow down to 64 MBit/s
        mu.write_register(SWB_READOUT_STATE_REGISTER_W, USE_GEN_LINK_setup);
    }

    // use stream merger for readout
    if (atoi(argv[1]) == 2) {
        uint32_t USE_GEN_LINK_setup = 0;
        uint32_t USE_STREAM_setup = 0;
        
        // readout only link 0 without merging
        // USE_GEN_LINK_setup = SET_USE_BIT_GEN_LINK(USE_GEN_LINK_setup);
        // USE_STREAM_setup = SET_USE_BIT_STREAM(USE_STREAM_setup);
        // mu.write_register(SWB_READOUT_STATE_REGISTER_W, USE_STREAM_setup);
        
        // setup datagenerator
        mu.write_register(DATAGENERATOR_DIVIDER_REGISTER_W, 0x0);//, 100);//3E8, 100);//3E8); // slow down to 64 MBit/s
        mu.write_register(SWB_READOUT_STATE_REGISTER_W, USE_GEN_LINK_setup);
    }

    // Enable all links (SC)
    mu.write_register_wait(FEB_ENABLE_REGISTER_W, 0xF, 100);
    // Enable all links (DATA)
    //mu.write_register_wait(DATA_LINK_MASK_REGISTER_2_W, 0xF, 100);
    //mu.write_register_wait(DATA_LINK_MASK_REGISTER_W, 0xFFFFFFFF, 100);
    mu.write_register_wait(DATA_LINK_MASK_REGISTER_W, 0xF, 100);
    // Enable only one link
    //mu.write_register_wait(DATA_LINK_MASK_REGISTER_W, 0x1, 100);

    // reset all
    mu.write_register_wait(RESET_REGISTER_W, 0x1, 100);
    sleep(2);
    mu.write_register_wait(RESET_REGISTER_W, 0x0, 100);

    if ( atoi(argv[1]) == 1 ) mu.write_register(0x13, 0x9);
    if ( atoi(argv[1]) == 2 ) mu.write_register(0x13, 0x3);

    mudaq::DmaMudaqDevice::DataBlock block;
    uint32_t newoffset;
    size_t read_words;

    uint32_t event_length = 0;
    uint32_t readindex = 0;
    uint32_t endofevent = 0;
    uint32_t lastendofevent = 0;
    uint32_t lastWritten = 0;
    int errno;
    uint64_t noData = 0;
    auto start_time = std::chrono::high_resolution_clock::now();

    for(int i=0; i < 8; i++)
        cout << hex << "0x" <<  dma_buf[i] << " ";
    cout << endl;

    if (atoi(argv[3]) == 1) {
        for(int i=0; i < 8; i++)
            cout << hex << "0x" <<  dma_buf[i+8] << " ";
        cout << endl;
        int cnt_loop = 0;
        // wait for requested data
        while ( (mu.read_register_ro(0x1C) & 1) == 0 ) {
            if ( cnt_loop == 1000 ) {
                cnt_loop = 0;
//                cout << mu.read_register_ro(0x1C) << endl;
            }
            cnt_loop = cnt_loop + 1;
        }
//        while(dma_buf[size/2/sizeof(uint32_t)-8] <= 0){

//         if (mu.last_written_addr() == 0) {
//             cout << "last_written" << endl;
//             continue;
//         }
//         if (mu.last_written_addr() == lastlastWritten) {
//             cout << "lastlast_written" << endl;
//             continue;
//         }
//         lastlastWritten = lastWritten;
//         lastWritten = mu.last_written_addr();
// 
//        myfile << "lastWritten" << endl;
//        for (int i = 0; i < 20; i++) {
//        char dma_buf_str[256];
//        sprintf(dma_buf_str, "%08X", dma_buf[lastWritten+i-20]);
//        myfile << lastWritten + i - 20 << "\t" << dma_buf_str << endl;
//        }
// 
//        myfile << "endofevent" << endl;
//         lastendofevent = endofevent;
//         endofevent = mu.last_endofevent_addr(); // now begin of event :)
// 
//         if ((endofevent+1)*8 > lastlastWritten) {
//             cout << "endofevent" << endl;
//             continue;
//         }
//         if ((dma_buf[(endofevent)*8-1] == 0xAFFEAFFE or dma_buf[(endofevent)*8-1] == 0x0000009c) && dma_buf[(endofevent)*8] == 0x1){
//             cout << hex << (endofevent+1)*8 << " " << lastWritten << " " << dma_buf[(endofevent+1)*8] << endl;
//         };
//        for (int i = 0; i < 20; i++) {
//        char dma_buf_str[256];
//        sprintf(dma_buf_str, "%08X", dma_buf[endofevent+i-20]);
//        myfile << endofevent + i - 20 << "\t" << dma_buf_str << endl;
//        }
  //      }
    }
    
    if ( atoi(argv[3]) != 1) {
        for ( int i = 0; i < 3; i++ ) {
            cout << "sleep " << i << "/3 s" << endl;
            sleep(i);
        }
    }

    cout << "start to write file" << endl;


//        if (readindex > 1000000) break;

//        lastWritten = mu.last_written_addr();

////        cout << "lastWritten" << hex << lastWritten << endl;
////        cout << "lastlastWritten" << hex << lastlastWritten << endl;


//        if (lastWritten == 0 || lastWritten == lastlastWritten ){
//            noData += 1;
//            continue;
//        }
//        if(lastlastWritten != 1){
//            for(int i=0; i < 8; i++)
//                cout << hex << "0x" <<  dma_buf[i] << " ";
//            cout << endl;
//        }
//        lastlastWritten = 1;

//        event_length = dma_buf[(readindex+7)%dma_buf_nwords];
//        if (event_length == 0) continue;

////        cout <<"length " << event_length << endl;
//        // do not overtake dma engine
//          if((readindex%dma_buf_nwords) > lastWritten){
//              if(dma_buf_nwords - (readindex % dma_buf_nwords) + lastWritten < event_length * 8 + 1){
////                  usleep(10);
//                  //cout<<"FE SLOW DOWN 1 index"<< (readindex%dma_buf_nwords) <<" lwr "<<lastWritten<<" eventL:"<<event_length<<" nWords "<<dma_buf_nwords<<endl;
//                  continue;
//              }
//          }else{
//              if(lastWritten - (readindex % dma_buf_nwords) < event_length * 8 + 1){
////                  usleep(10);
//                  //cout<<"FE SLOW DOWN 2 index"<< (readindex%dma_buf_nwords) <<" lwr "<<lastWritten<<" eventL:"<<event_length<<" nWords "<<dma_buf_nwords<<endl;
//                  continue;
//              }
//          }

////          auto current_time = std::chrono::high_resolution_clock::now();
////          auto time = current_time - start_time;
////          if(std::chrono::duration_cast<std::chrono::microseconds>(time).count() >= 10000)// 3.6e+9)
////              break;







////        errno = mu.read_block(block, dma_buf);
////        if(errno == mudaq::DmaMudaqDevice::READ_SUCCESS){
////            /* Extract # of words read, set new position in ring buffer */

////            newoffset = block.give_offset();
////            read_words += block.size();

////            auto current_time = std::chrono::high_resolution_clock::now();
////            auto time = current_time - start_time;
////            if(std::chrono::duration_cast<std::chrono::microseconds>(time).count() >= 100000)// 3.6e+9)
////                break;
////        }
////        else if(errno == mudaq::DmaMudaqDevice::READ_NODATA){
////            noData += 1;
////            continue;
////        }
////        else {
////            cout << "DMA Read error " << errno << endl;
////            break;
////        }

////    cout << "No data: " << noData << endl;

//    uint64_t lastmemaddr = mu.last_written_addr();

////    cout << "lastmemaddr is " << hex << lastmemaddr << endl;

////    cout << "Writing file!" << endl;

////    int firstindex = -1;
////    int lastindex = -1;
////    for(uint64_t i = 0; i < lastmemaddr; i++){
////        char dma_buf_str[256];
////        sprintf(dma_buf_str, "%08X", dma_buf[i]);
////        myfile << i << "\t" << dma_buf_str  << endl;
////        if(dma_buf[i] != 0){
////            if(firstindex < 0)
////                firstindex = i;
////        lastindex = i;
////        }
////    }
//    }

    mu.disable();
    // stop generator
    if (atoi(argv[1]) == 1) {
        uint32_t datagen_setup = 0;
        datagen_setup = UNSET_DATAGENERATOR_BIT_ENABLE(datagen_setup);
        mu.write_register_wait(DATAGENERATOR_REGISTER_W, datagen_setup, 100);
        mu.write_register_wait(DATAGENERATOR_DIVIDER_REGISTER_W, 0x0, 100);
    }

    // reset all
    mu.write_register_wait(RESET_REGISTER_W, 0x1, 100);

    char dma_buf_str[256];
    for (int j = 0 ; j < size/sizeof(uint32_t); j++){
        if(j % (1024*1024) == 0) printf("j = %d\n", j);
        sprintf(dma_buf_str, "%08X", dma_buf[j]);
        myfile << j << "\t" << dma_buf_str << endl;
    }

    // stop generator
//    datagen_setup = UNSET_DATAGENERATOR_BIT_ENABLE(datagen_setup);
//    mu.write_register_wait(DATAGENERATOR_REGISTER_W, datagen_setup, 100);
//    mu.write_register_wait(DMA_SLOW_DOWN_REGISTER_W, 0x3E8, 100);//3E8); // slow down to 64 MBit/s

    mu.close();

    myfile.close();
    return 0;
}
