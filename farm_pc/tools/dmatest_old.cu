#include "hip/hip_runtime.h"
/**
 * open a mudaq device and measure DMA speed
 * use data generator from counter with 250 MHz clock
 *
 * @author      Dorothea vom Bruch <vombruch@physi.uni-heidelberg.de>
 *              adapted from code by Fabian Foerster and Qinhua Huang
 * @date        2015-01-22
 */

#include <iostream>
#include <unistd.h>
#include <chrono>
#include <stdio.h>
#include <sstream>
#include <limits>
#include <fstream>
#include <sys/mman.h>
#include <chrono>
#include <stdlib.h>
#include <cassert>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#include "mudaq_device.h"

#define DMA_SLOW_DOWN_REGISTER_W		0x06

using namespace std;

int main(int argc, char *argv[])
{

    ofstream myfile;
    myfile.open("memory_content.txt");
    if ( !myfile ) {
      cout << "Could not open file " << endl;
      return -1;
    }

    myfile << "idx" << "\t" << "data" << "\t" << "event_length" << endl;

    system("echo machmalkeins | sudo -S /home/labor/daq/driver/compactify.sh");
    usleep(1000000);
    system("echo machmalkeins | sudo -S /home/labor/daq/driver/compactify.sh");
    usleep(1000000);

    size_t dma_buf_size = MUDAQ_DMABUF_DATA_LEN;
    volatile uint32_t *dma_buf;
    size_t size = MUDAQ_DMABUF_DATA_LEN;
    uint32_t dma_buf_nwords = dma_buf_size/sizeof(uint32_t);

    if(hipHostMalloc( (void**)&dma_buf, size ) != hipSuccess){
        cout << "Allocation failed!" << endl;
        return -1;
    }

    // initialize to zero
    for (int i = 0; i <  size/sizeof(uint32_t) ; i++) {
      (dma_buf)[ i ] = 0;

    }

    // Host memory
    uint32_t * cpu_mem = (uint32_t *)malloc(size);
    if(!cpu_mem){
        cout << "CPU memory allocation failed" << endl;
        return -1;
    }


    /* Open mudaq device */
    mudaq::DmaMudaqDevice mu("/dev/mudaq0");
    if ( !mu.open() ) {
        cout << "Could not open device " << endl;
        return -1;
    }

    if ( !mu.is_ok() ) return -1;
    cout << "MuDaq is ok" << endl;

    struct mesg user_message;
    user_message.address = dma_buf;
    user_message.size = size;

    /* map memory to bus addresses for FPGA */
    int ret_val = mu.map_pinned_dma_mem( user_message );

    if ( ret_val < 0 ) {
        cout << "Mapping failed " << endl;
        mu.disable();
        mu.close();
        free( (void *)dma_buf );
        return ret_val;
    }

    // Set up data generator
    uint32_t datagen_setup = 0;
    mu.write_register_wait(DMA_SLOW_DOWN_REGISTER_W, 0x3E8, 100);//3E8); // slow down to 64 MBit/s
    datagen_setup = SET_DATAGENERATOR_BIT_ENABLE_PIXEL(datagen_setup);
    mu.write_register_wait(DATAGENERATOR_REGISTER_W, datagen_setup, 100);

    // reset all
    uint32_t reset_reg = 0;
    reset_reg = SET_RESET_BIT_ALL(reset_reg);
    mu.write_register_wait(RESET_REGISTER_W, reset_reg, 100);

    // Enable register on FPGA for continous readout and enable dma
    uint32_t lastlastWritten = mu.last_written_addr();
    mu.enable_continous_readout(0);
    usleep(10);
    mu.write_register_wait(RESET_REGISTER_W, 0x0, 100);

    mudaq::DmaMudaqDevice::DataBlock block;
    uint32_t newoffset;
    size_t read_words;

    uint32_t event_length = 0;
    uint32_t readindex = 0;
    uint32_t lastWritten = 0;
    int errno;
    uint64_t noData = 0;
    auto start_time = std::chrono::high_resolution_clock::now();

    for(int i=0; i < 8; i++)
        cout << hex << "0x" <<  dma_buf[i] << " ";
    cout << endl;

    while(true){


        if (readindex > 1000000) break;

        lastWritten = mu.last_written_addr();

//        cout << "lastWritten" << hex << lastWritten << endl;
//        cout << "lastlastWritten" << hex << lastlastWritten << endl;


        if (lastWritten == 0 || lastWritten == lastlastWritten ){
            noData += 1;
            continue;
        }
        if(lastlastWritten != 1){
            for(int i=0; i < 8; i++)
                cout << hex << "0x" <<  dma_buf[i] << " ";
            cout << endl;
        }

        lastlastWritten = 1;

        event_length = dma_buf[(readindex+7)%dma_buf_nwords];
        if (event_length == 0) continue;

        cout <<"length " << event_length << endl;
        // do not overtake dma engine
          if((readindex%dma_buf_nwords) > lastWritten){
              if(dma_buf_nwords - (readindex % dma_buf_nwords) + lastWritten < event_length * 8 + 1){
//                  usleep(10);
                  //cout<<"FE SLOW DOWN 1 index"<< (readindex%dma_buf_nwords) <<" lwr "<<lastWritten<<" eventL:"<<event_length<<" nWords "<<dma_buf_nwords<<endl;
                  continue;
              }
          }else{
              if(lastWritten - (readindex % dma_buf_nwords) < event_length * 8 + 1){
//                  usleep(10);
                  //cout<<"FE SLOW DOWN 2 index"<< (readindex%dma_buf_nwords) <<" lwr "<<lastWritten<<" eventL:"<<event_length<<" nWords "<<dma_buf_nwords<<endl;
                  continue;
              }
          }

//          auto current_time = std::chrono::high_resolution_clock::now();
//          auto time = current_time - start_time;
//          if(std::chrono::duration_cast<std::chrono::microseconds>(time).count() >= 10000)// 3.6e+9)
//              break;



        for (int j = 0 ; j < event_length; j++){
            char dma_buf_str[256];
            sprintf(dma_buf_str, "%08X", dma_buf[(readindex + 6)%dma_buf_nwords]);
            myfile << readindex + 6 << "\t" << dma_buf_str << "\t" << event_length  << endl;
            readindex = readindex + 8;
        }



//        errno = mu.read_block(block, dma_buf);
//        if(errno == mudaq::DmaMudaqDevice::READ_SUCCESS){
//            /* Extract # of words read, set new position in ring buffer */

//            newoffset = block.give_offset();
//            read_words += block.size();

//            auto current_time = std::chrono::high_resolution_clock::now();
//            auto time = current_time - start_time;
//            if(std::chrono::duration_cast<std::chrono::microseconds>(time).count() >= 100000)// 3.6e+9)
//                break;
//        }
//        else if(errno == mudaq::DmaMudaqDevice::READ_NODATA){
//            noData += 1;
//            continue;
//        }
//        else {
//            cout << "DMA Read error " << errno << endl;
//            break;
//        }

//    cout << "No data: " << noData << endl;

    uint64_t lastmemaddr = mu.last_written_addr();

    cout << "lastmemaddr is " << hex << lastmemaddr << endl;

    cout << "Writing file!" << endl;

//    int firstindex = -1;
//    int lastindex = -1;
//    for(uint64_t i = 0; i < lastmemaddr; i++){
//        char dma_buf_str[256];
//        sprintf(dma_buf_str, "%08X", dma_buf[i]);
//        myfile << i << "\t" << dma_buf_str  << endl;
//        if(dma_buf[i] != 0){
//            if(firstindex < 0)
//                firstindex = i;
//        lastindex = i;
//        }
//    }
    }

    // stop generator
    datagen_setup = UNSET_DATAGENERATOR_BIT_ENABLE(datagen_setup);
    mu.write_register_wait(DATAGENERATOR_REGISTER_W, datagen_setup, 100);
    mu.write_register_wait(DMA_SLOW_DOWN_REGISTER_W, 0x3E8, 100);//3E8); // slow down to 64 MBit/s

    mu.disable();
    mu.close();

    myfile.close();
    return 0;
}
